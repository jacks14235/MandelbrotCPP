#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "CImg.h"
#include ""
#include <chrono> 

using namespace std::chrono;

using namespace cimg_library;

__global__ void mandel(double *real, double *imag, int *out, int n, int maxIt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	double mReal = real[i];
	double mImag = imag[i];
	for (int iter = 0; iter < maxIt; iter++) {
		double mReal2 = mReal * mReal;
		double mImag2 = mImag * mImag;
		double nReal = mReal;
		mReal = mReal2 - mImag2 + real[i];
		mImag = 2 * nReal * mImag + imag[i];
		if (mReal2 + mImag2 > 4) {
			out[i] = iter;
			//charOut[i] = black;
			return;
		}
	}
	out[i] = maxIt;
	//charOut[i] = white;
	
}


int* makeImage(const int xRes, const int yRes, double xStart, double xEnd, double  yStart, double yEnd, int maxIt) {
	double *real;
	double *imag;
	int *out;
	//unsigned char **charOut;

	hipMallocManaged(&real, xRes * yRes * sizeof(double));
	hipMallocManaged(&imag, xRes * yRes * sizeof(double));
	hipMallocManaged(&out, xRes * yRes * sizeof(int));

	double xStep = (xEnd - xStart) / (double)xRes;
	double yStep = (yEnd - yStart) / (double)yRes;
	//std::cout << xStep;
	//std::cout << yStep;
	int c = 0;
	for (int i = 0; i < xRes; i++) {
		for (int j = 0; j < yRes; j++) {
			real[c] = xStart + i * xStep;
			imag[c] = yStart + j * yStep;
			out[i] = 0;
			c++;
		}
	}
	auto start = high_resolution_clock::now();

	mandel << <xRes * yRes / 1024, 1024 >> > (real, imag, out, xRes * yRes, maxIt);

	hipDeviceSynchronize();
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	std::cout << duration.count();
	std::cout << "\n\n\n";

	hipFree(real);
	hipFree(imag);
	return out;
}